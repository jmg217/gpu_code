#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <ctime>
#include <cmath>
#include "enum_header.h"
#include <unistd.h>
#include <stdio.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

double* three_dim_index(double* matrix, int i, int j, int k, double m, int b, int num_assets);

__device__ double* two_dim_indexGPU(double* vector, int i, int j, double m, int b);

__device__ double* three_dim_indexGPU(double* matrix, int i, int j, int k, double m, int b, int num_assets);


__global__ void init(unsigned int seed, hiprandState_t* states);

__global__ void MeshGenKernel(double* X_device, double* delta_device, double* sigma_device,double* X0_device, int N, double strike, double r, double delta_t, int b, double m, int num_assets, hiprandState_t* states, double* asset_amount_device){

int idx =blockDim.x*blockIdx.x + threadIdx.x;

if(idx<N){
double Xi;
int m_int=(int)m;
for(int i=0; i<m_int; i++){
        if(i==0){
        
                for(int ll=0; ll<num_assets; ll++){
                       // Z=distribution(generator);
			Z=hiprand_normal_double(&states[idx])
        		*three_dim_indexGPU(X_device, i, idx, ll, m, b) = X0_device[ll] +  (r-delta_device[ll]-0.5*pow(sigma_device[ll], 2))*delta_t + sigma_device[ll]*sqrt(delta_t)*Z;

                }
        
        }

        if(i>0){
                for(int jj=0; jj<num_assets; jj++){
                        //Z=distribution(generator);
			Z=hiprand_normal_double(&states[idx])
                        Xi=*three_dim_indexGPU(X_device, (i-1), idx, jj, m, b);
                        *three_dim_indexGPU(X_device, i, idx, jj, m, b)=Xi +  (r-delta_device[jj]-0.5*pow(sigma_device[jj], 2))*delta_t + sigma_device[jj]*sqrt(delta_t)*Z;
                }
        
        }
}

}
}

 
void mesh_generation(int b, int num_assets, double m, double X0[], double sigma[], double delta[], double asset_amount[], double* X, double strike, double r, double delta_t, int num_assets){
int N= b;
int m_int=(int)m;
double* X0_host;
X0_host =X0;

double* sigma_host;
sigma_host =sigma;

double* delta_host;
delta_host =delta;

double* asset_amount_host;
asset_amount_host =asset_amount;

int X_N=(m_int) * b * (num_assets);
int delta_N= num_assets;
int sigma_N=num_assets;
int X0_N=num_assets;
int asset_amount_N = num_assets;

double* X_device;
double* sigma_device;
double* delta_device;
double* X0_device;
double* asset_amount_device;

hipMalloc((void**) &X_device, X_N*sizeof(double) );
hipMemcpy(X_device, X, X_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &X0_device, X0_N*sizeof(double) );
hipMemcpy(X0_device, X0_host, X0_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &sigma_device, sigma_N*sizeof(double) );
hipMemcpy(sigma_device, sigma_host, sigma_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &delta_device, delta_N*sizeof(double) );
hipMemcpy(delta_device, delta_host, delta_N*sizeof(double), hipMemcpyHostToDevice);

hipMalloc((void**) &asset_amount_device, asset_amount_N*sizeof(double) );
hipMemcpy(asset_amount_device, asset_amount_host, asset_amount_N*sizeof(double), hipMemcpyHostToDevice);

dim3 gridDim((int)ceil(N/512.0));
dim3 blockDim(512.0);

hiprandState_t* states;

hipMalloc((void**) &states, N * sizeof(hiprandState_t));

init<<<gridDim, blockDim>>>(time(0), states);

hipDeviceSynchronize();

MeshGenKernel<<<gridDim, blockDim>>>(X_device, delta_device, sigma_device, X0_device, N, strike, r, delta_t, b,  m, num_assets, states, asset_amount_device);

hipDeviceSynchronize();

hipMemcpy(X, X_device, sizeof(double)*X_N, hipMemcpyDeviceToHost);

hipFree(X_device);
hipFree(sigma_device);
hipFree(delta_device);
hipFree(X0_device);
hipFree(asset_amount_device);

hipDeviceReset();
}

